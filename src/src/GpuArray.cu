#include "GpuArray.h"
#include <iostream>

void GpuArray_allocateDevice_Cuda(GpuArray *gpuArray) {
    hipMalloc((void **) &gpuArray->deviceData,
               sizeof(float) * gpuArray->size
    );
}

void GpuArray_deallocateDevice_Cuda(GpuArray *gpuArray) {
    hipFree(gpuArray->deviceData);
}


void GpuArray_toHost_Cuda(GpuArray *gpuArray) {
    hipMemcpy(
            *(gpuArray->data),
            gpuArray->deviceData,
            sizeof(float) * gpuArray->size,
            hipMemcpyDeviceToHost
    );
}

void GpuArray_toDevice_Cuda(GpuArray *gpuArray) {
    hipMemcpy(
            gpuArray->deviceData,
            *(gpuArray->data),
            sizeof(float) * gpuArray->size,
            hipMemcpyHostToDevice
    );
}


