#include "hip/hip_runtime.h"
#include "GpuArray.h"
#include <iostream>

__global__ void sumReductionKernel(
        float *a,
        float *r,
        int size
) {
    __shared__ float sharedData[1024];

    // Thread index
    unsigned int tid = threadIdx.x;

    // Global index
    unsigned int index = blockIdx.x * blockDim.x + tid;

    unsigned int stride;

    if (index < size) {
        sharedData[tid] = a[index];
    }
    else {
        sharedData[tid] = 0.0;
    }
    __syncthreads();
    for (stride = blockDim.x / 2; stride >0; stride >>=1) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }
    if (tid == 0) {
        r[blockIdx.x] = sharedData[0];
    }
}

void GpuArray_allocateDevice_Cuda(GpuArray *gpuArray) {
    hipMalloc((void **) &gpuArray->deviceData,
               sizeof(float) * gpuArray->size
    );
}

void GpuArray_deallocateDevice_Cuda(GpuArray *gpuArray) {
    hipFree(gpuArray->deviceData);
}


void GpuArray_toHost_Cuda(GpuArray *gpuArray) {
    hipMemcpy(
            *(gpuArray->data),
            gpuArray->deviceData,
            sizeof(float) * gpuArray->size,
            hipMemcpyDeviceToHost
    );
}

void GpuArray_toDevice_Cuda(GpuArray *gpuArray) {
    hipMemcpy(
            gpuArray->deviceData,
            *(gpuArray->data),
            sizeof(float) * gpuArray->size,
            hipMemcpyHostToDevice
    );
}

float GpuArray_sumReduction_Cuda(GpuArray *gpuArray) {
    int numThreads = 1024;
    int numBlocks, i;
    GpuArray *rArray;
    float r;
    numBlocks = (gpuArray->size / numThreads) + 1;
    rArray = new GpuArray;
    rArray->allocate(&numBlocks);
    rArray->allocateDevice();
    sumReductionKernel<<<numBlocks, numThreads>>>(
            gpuArray->deviceData,
            rArray->deviceData,
            gpuArray->size

    );
    rArray->toHost();
    r = 0.0;
    for (i = 0; i <numBlocks; i++) {
        r += (*(rArray->data))[i];
    }
    rArray->deallocateDevice();
    rArray->deallocate();
    delete rArray;
    return r;
}

